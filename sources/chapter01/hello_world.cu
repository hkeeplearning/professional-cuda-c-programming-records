#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void HelloWorldGpu()
{
    printf("Hello world from GPU thread %d!\n", threadIdx.x);
}

extern "C" void HelloWorld()
{
    HelloWorldGpu<<<1, 10>>>();
    hipDeviceReset(); // 释放和清空资源，同时也起到同步的作用
    // cudaDeviceSynchronize();
    return;
}